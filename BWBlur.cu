#include "hip/hip_runtime.h"
#include "BWBlur.h"
#include<math.h>
__global__ void BWBlur(uchar4* image, uchar4* out, size_t height, size_t width) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x < height && y < width) {
		int idx = x * width + y;

		int idx = x * width + y;

		image[idx].x = 255 - out[idx].x;
		image[idx].y = 255 - out[idx].y;
		image[idx].z = 255 - out[idx].z;


		/*int count = 0;
		int sum = 0;
		
		for (int i = y - 1; i <= y + 1; i++)
		{
			for (int j = x - 1; j <= x + 1; j++)
			{
				 int ind = x * width + y;
				if (i >= 0 && i < width && j >= 0 && j < height)
				{
					count+=1;
					sum = sum + image[ind].x;
				}
				
			}
		}
		int val = (sum / count);
		out[idx].x = val;
		out[idx].y = val;
		out[idx].z = val;*/

		
	}
}

void executeBWBlur(uchar4* image, size_t height, size_t width, float gc) {

	uchar4* d_image;
	uchar4* d_out;
	hipMalloc(&d_image, height * width * sizeof(uchar4));
	hipMalloc(&d_out, height * width * sizeof(uchar4));
	hipMemcpy(d_image, image, height * width * sizeof(uchar4), hipMemcpyHostToDevice);
	BWBlur <<<dim3(1 + ((height - 1) / 32), 1 + ((width - 1) / 32), 1), dim3(32, 32, 1) >>> (d_image, d_out, height, width);
	hipMemcpy(image, d_out, height * width * sizeof(uchar4), hipMemcpyDeviceToHost);

}