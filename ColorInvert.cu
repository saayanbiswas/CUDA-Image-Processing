#include "hip/hip_runtime.h"
#include "ColorInvert.h"
#include<stdlib.h>
__constant__ int mask[3 * 3];

__global__ void ColorInvert(uchar4* image, uchar4* out, size_t height, size_t width) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x < height && y < width) 
	{
		int idx = x * width + y;

		int start_r = x - 1;
		int start_c = y - 1;

		int tempr = 0;
		int tempb = 0;
		int tempg = 0;
		for (int i = 0; i < 3; i++)
		{
			for (int j = 0; j < 3; j++)
			{
				if ((start_r + i) >= 0 && (start_r + i) < height) 
				{
					if ((start_c + j) >= 0 && (start_c + j) < width)
					{
						tempr += image[(start_r + i) * width + (start_c + j)].x * mask[i * 3 + j];
						tempg += image[(start_r + i) * width + (start_c + j)].y * mask[i * 3 + j];
						tempb += image[(start_r + i) * width + (start_c + j)].z * mask[i * 3 + j];
					}
				}
			}
		}
		
		int div = 9;
		/*out[idx].x = (tempr / div > 255) ? 255:(tempr / div);
		out[idx].y = (tempg / div > 255) ? 255: tempg / div;
		out[idx].z = (tempb / div > 255) ? 255: tempb / div;*/
		out[idx].x = image[idx].x + tempr / div;
		out[idx].y = image[idx].y + tempg / div;
		out[idx].z = image[idx].z + tempb / div;


	}
}

void executeColorInvert(uchar4* image, size_t height, size_t width) {

	uchar4* d_image;
	uchar4* d_out;
	int filter[]= {	-1,-1,-1,
					-1,9,-1,
					-1,-1,-1};
	size_t bytes_m = 3 * 3 * sizeof(int);
	hipMalloc(&d_image, height * width * sizeof(uchar4));
	hipMalloc(&d_out, height * width * sizeof(uchar4));
	hipMemcpyToSymbol(HIP_SYMBOL(mask), filter, bytes_m);
	hipMemcpy(d_image, image, height * width * sizeof(uchar4), hipMemcpyHostToDevice);
	ColorInvert <<<dim3(1 + ((height - 1) / 32), 1 + ((width - 1) / 32), 1), dim3(32, 32, 1) >>> (d_image,d_out, height, width);
	hipMemcpy(image, d_out, height * width * sizeof(uchar4), hipMemcpyDeviceToHost);

}