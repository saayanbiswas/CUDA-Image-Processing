#include "hip/hip_runtime.h"
#include "GammaCorrection.h"
#include<math.h>
__global__ void GammaCorrection(uchar4* image, size_t height, size_t width, float gc) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x < height && y < width) {
		int idx = x * width + y;

		float r, g, b;
		r=image[idx].x ;
		g=image[idx].y  ;
		b=image[idx].z  ;
		r = 255 * pow((r / 255),(1/gc));
		g = 255 * pow((g / 255),(1/gc));
		b = 255 * pow((b / 255),(1/gc));
		image[idx].x = r;
		image[idx].y = g;
		image[idx].z = b;
	}
}

void executeGammaCorrection(uchar4* image, size_t height, size_t width, float gc) {

	uchar4* d_image;
	hipMalloc(&d_image, height * width * sizeof(uchar4));
	hipMemcpy(d_image, image, height * width * sizeof(uchar4), hipMemcpyHostToDevice);
	GammaCorrection <<<dim3(1 + ((height - 1) / 32), 1 + ((width - 1) / 32), 1), dim3(32, 32, 1) >> > (d_image, height, width,gc);
	hipMemcpy(image, d_image, height * width * sizeof(uchar4), hipMemcpyDeviceToHost);

}