#include "hip/hip_runtime.h"
#include "Smoothen.h"

__global__ void Smoothen(uchar4* image, size_t height, size_t width) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x < height && y < width) {
		int idx = x * width + y;
		unsigned char val = 0.299 * image[idx].x + 0.587 * image[idx].y + 0.114 * image[idx].z;
		image[idx].x = val;
		image[idx].y = val;
		image[idx].z = val;
	}
}

void executeSmoothen(uchar4* image, size_t height, size_t width) {

	uchar4* d_image;
	hipMalloc(&d_image, height * width * sizeof(uchar4));
	hipMemcpy(d_image, image, height * width * sizeof(uchar4), hipMemcpyHostToDevice);
	Smoothen <<<dim3(1 + ((height - 1) / 32), 1 + ((width - 1) / 32), 1), dim3(32, 32, 1) >>> (d_image, height, width);
	hipMemcpy(image, d_image, height * width * sizeof(uchar4), hipMemcpyDeviceToHost);

}